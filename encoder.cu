#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <hip/hip_runtime.h>

#define BMP_HEADER_SIZE 54

__device__ uint8_t secret_key[3] = { 0xAB, 0xCD, 0xEF };

__global__ void encode_image(uint8_t* image, uint32_t width, uint32_t height)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = idx * 3 + idy * width * 3;
    if (idx < width && idy < height) {
        uint8_t r = image[offset];
        uint8_t g = image[offset + 1];
        uint8_t b = image[offset + 2];
        image[offset] = r ^ secret_key[0];
        image[offset + 1] = g ^ secret_key[1];
        image[offset + 2] = b ^ secret_key[2];
    }
}

int main(int argc, char** argv)
{
    if (argc != 2) {
        printf("Usage: %s input_image.bmp\n", argv[0]);
        return 1;
    }

    // Open input image file
    FILE* fp = fopen(argv[1], "rb");
    if (fp == NULL) {
        printf("Failed to open input image file\n");
        return 1;
    }

    // Read BMP header
    uint8_t header[BMP_HEADER_SIZE];
    if (fread(header, 1, BMP_HEADER_SIZE, fp) != BMP_HEADER_SIZE) {
        printf("Failed to read BMP header\n");
        return 1;
    }

    // Read image data
    uint32_t width = *(uint32_t*)(header + 18);
    uint32_t height = *(uint32_t*)(header + 22);
    uint32_t data_size = *(uint32_t*)(header + 34);
    uint8_t* image = (uint8_t*)malloc(data_size);
    if (fread(image, 1, data_size, fp) != data_size) {
        printf("Failed to read image data\n");
        return 1;
    }

    // Close input image file
    fclose(fp);

    // Allocate memory on GPU
    uint8_t* d_image;
    hipMalloc((void**)&d_image, data_size);

    // Copy image data from CPU to GPU
    hipMemcpy(d_image, image, data_size, hipMemcpyHostToDevice);

    // Define block size and grid size
    dim3 block_size(16, 16);
    dim3 grid_size((width + block_size.x - 1) / block_size.x, (height + block_size.y - 1) / block_size.y);

    // Launch kernel to encode image
    encode_image<<<grid_size, block_size>>>(d_image, width, height);

    // Copy encoded image data from GPU to CPU
    hipMemcpy(image, d_image, data_size, hipMemcpyDeviceToHost);

    // Open output image file
    fp = fopen("encoded_image.bmp", "wb");
    if (fp == NULL) {
        printf("Failed to open output image file\n");
        return 1;
    }

    // Write BMP header
    if (fwrite(header, 1, BMP_HEADER_SIZE, fp) != BMP_HEADER_SIZE) {
        printf("Failed to write BMP header\n");
        return 1;
    }

    // Write encoded image data
    if (fwrite(image, 1, data_size, fp) != data_size) {
        printf("Failed to write encoded image data\n");
        return 1;
    }

    // Close output image file
    fclose(fp);

    // Free memory on GPU and CPU
    hipFree(d_image);
    free(image);

    return 0;
}
